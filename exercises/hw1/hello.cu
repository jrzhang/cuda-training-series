
#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void hello(){

  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main(){
  hello<<<2, 2>>>();
  gpuErrchk( hipPeekAtLastError() );
  
  gpuErrchk(hipDeviceSynchronize());
}

